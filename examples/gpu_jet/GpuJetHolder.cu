#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "GpuJetHolder.h"
#include <fmt/format.h>



namespace ceres::examples::internal
{

__global__ void Kernel( const GpuJetHolder::ScalarType* pData , GpuJetHolder::ScalarType* derive , unsigned int NumPoints,GpuJetHolder::JetT* pJets,unsigned int pperthread,VectorScalarCostFunctor Functor )
{
    const unsigned int currThreadIdx = threadIdx.x + blockDim.x * blockIdx.x;
    if (currThreadIdx * pperthread >= NumPoints + pperthread)
    {
        return;
    }
    for(unsigned int i=0;i<pperthread;i++)
    {
        unsigned int pIdx = currThreadIdx*pperthread+i;
        if (pIdx >= NumPoints)
        {
            return;
        }
        GpuJetHolder::JetT* jetArg = pJets + pIdx * 2;
        jetArg[ 0 ] = GpuJetHolder::JetT( pData[ pIdx * 2 ] , 0 );
        jetArg[ 1 ] = GpuJetHolder::JetT( pData[ pIdx * 2 + 1 ] , 1 );
        GpuJetHolder::JetT res;
        Functor( jetArg , &res );
        derive[ pIdx * 2 ] = res.v[ 0 ];
        derive[ pIdx * 2 +1] = res.v[ 1 ];
    }
    
    
}

void GpuJetHolder::RunInternalGPUWithSettings(clock_t& gpu_dur,unsigned int pperthread,unsigned int NumThreadsInBlock)
{
    memset( _derives.get( ) , 0 , _points_num * sizeof( decltype( std::declval<JetT>( ).a ) ) * 2 );
    auto gpu_start = clock( );
    _devPoints->CopyFromCpu( _points.get( ) , 2 * _points_num );
    const int NumBlocks = ( _points_num / (NumThreadsInBlock*pperthread) ) + (_points_num % (NumThreadsInBlock*pperthread) == 0 ? 0 : 1);
    Kernel << <NumBlocks , NumThreadsInBlock >> > ( _devPoints->data( ) , _devDerives->data( ) , _points_num,_pCudaBuffer->data(),pperthread,VectorScalarCostFunctor() );
    auto res = ::hipDeviceSynchronize( );
    if (res != hipError_t::hipSuccess)
    {
        auto err_str = fmt::format( "error {} \"{}\"" , res , ::hipGetErrorString( res ) );
        throw std::runtime_error( err_str );
    }
    _devDerives->CopyToCpu( _derives.get( ) , _devDerives->size( ) );
    gpu_dur = clock( ) - gpu_start;
    //return DeriveMatrix::Map( _derives.get( ) , _points_num , 2 ).eval( );
    //for (size_t i = 0; i < _points_num; i++)
    //{
    //    float analytical = ScalarScalarCostFunctor::analyticalDeriv( _points[ 2 * i ] );
    //    fmt::print( " x {} deriv  {} analytical cpu {} diff {} \n" , _points[ 2 * i ] , _derives[ 2 * i ] , analytical , abs( analytical - _derives[ 2 * i ] ) );
    //}
}

}